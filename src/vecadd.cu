#include "hip/hip_runtime.h"
#include <iostream>

//using namespace std;

//获取GPU设备信息
void getDeviceInfo(){
	int dev = 0;
	hipDeviceProp_t devProp;
	hipGetDeviceProperties(&devProp, dev);
	std::cout << "使用GPU device " << dev << ": " << devProp.name << std::endl;
//	std::cout << "SP的数量：" << devProp. << std::endl;
	std::cout << "SM的数量：" << devProp.multiProcessorCount << std::endl;
	std::cout << "每个线程块的共享内存大小：" << devProp.sharedMemPerBlock / 1024.0 << " KB" << std::endl;
	std::cout << "每个线程块的最大线程数：" << devProp.maxThreadsPerBlock << std::endl;
	std::cout << "每个SM的最大线程数：" << devProp.maxThreadsPerMultiProcessor << std::endl;
	std::cout << "每个SM的最大线程束数：" << devProp.maxThreadsPerMultiProcessor / 32 << std::endl;
}
// 两个向量加法kernel，grid和block均为一维
__global__ void vecAdd(float* x, float * y, float* z, int n)
{
    // 获取全局索引
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    // 步长
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride)
    {
        z[i] = x[i] + y[i];
    }
}
int testVecAdd(){
	int N = 1 << 20;
	int nBytes = N * sizeof(float);
	// 申请host内存
	float *x, *y, *z;
	x = (float*)malloc(nBytes);
	y = (float*)malloc(nBytes);
	z = (float*)malloc(nBytes);

	// 初始化数据
	for (int i = 0; i < N; i++)
	{
		x[i] = 1.0 * i;
		y[i] = 2.0 * i;
	}

	// 申请device内存
	float *d_x, *d_y, *d_z;
	hipMalloc((void**)&d_x, nBytes);
	hipMalloc((void**)&d_y, nBytes);
	hipMalloc((void**)&d_z, nBytes);

	// 将host数据拷贝到device
	hipMemcpy((void*)d_x, (void*)x, nBytes, hipMemcpyHostToDevice);
	hipMemcpy((void*)d_y, (void*)y, nBytes, hipMemcpyHostToDevice);
	// 定义kernel的执行配置
	dim3 blockSize(256);
	dim3 gridSize((N + blockSize.x - 1) / blockSize.x);
	// 执行kernel
	vecAdd<<<gridSize, blockSize>>>(d_x,d_y,d_z,N);

	hipMemcpy((void*)z,(void*)d_z,nBytes,hipMemcpyDeviceToHost);

	for(int i=0;i < 10;i++){
		std::cout << z[i] << " ";
	}
	hipFree(d_x);
	hipFree(d_y);
	hipFree(d_z);
	free(x);
	free(y);
	free(z);

	return 0;
}

int testMallocManager(){
	int N = 1 << 20;
	int nBytes = N * sizeof(float);

	// 申请托管内存
	float *x, *y, *z;
	hipMallocManaged((void**)&x, nBytes);
	hipMallocManaged((void**)&y, nBytes);
	hipMallocManaged((void**)&z, nBytes);

	// 初始化数据
	for (int i = 0; i < N; ++i)
	{
		x[i] = 1.0 * i;
		y[i] = 2.0 * i;
	}

	// 定义kernel的执行配置
	dim3 blockSize(256);
	dim3 gridSize((N + blockSize.x - 1) / blockSize.x);
	// 执行kernel
	vecAdd << < gridSize, blockSize >> >(x, y, z, N);

	// 同步device 保证结果能正确访问
	hipDeviceSynchronize();
	// 检查执行结果
	for(int i=0;i < 10;i++){
		std::cout << z[i] << " ";
	}
	// 释放内存
	hipFree(x);
	hipFree(y);
	hipFree(z);

	return 0;
}



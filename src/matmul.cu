#include<iostream>
#include<hip/device_functions.h>
#include<hip/hip_runtime.h>

// Thread block size
#define BLOCK_SIZE 16
//using namespace std;

// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.stride + col)
typedef struct {
    int width;
    int height;
    int stride;
    float* elements;
} Matrix;

// Get a matrix element
__device__ float GetElement(const Matrix A, int row, int col)
{
    return A.elements[row * A.stride + col];
}

// Set a matrix element
__device__ void SetElement(Matrix A, int row, int col,
                           float value)
{
    A.elements[row * A.stride + col] = value;
}


// Get the BLOCK_SIZExBLOCK_SIZE sub-matrix Asub of A that is
// located col sub-matrices to the right and row sub-matrices down
// from the upper-left corner of A
 __device__ Matrix GetSubMatrix(Matrix A, int row, int col)
{
    Matrix Asub;
    Asub.width    = BLOCK_SIZE;
    Asub.height   = BLOCK_SIZE;
    Asub.stride   = A.stride;
    Asub.elements = &A.elements[A.stride * BLOCK_SIZE * row
                                         + BLOCK_SIZE * col];
    return Asub;
}

 // Forward declaration of the matrix multiplication kernel
 __global__ void MatMulKernel(const Matrix, const Matrix, Matrix);


 // Matrix multiplication - Host code
 // Matrix dimensions are assumed to be multiples of BLOCK_SIZE
 void MatMul(const Matrix A, const Matrix B, Matrix C)
 {
     // Load A and B to device memory
     Matrix d_A;
     d_A.width = d_A.stride = A.width; d_A.height = A.height;
     size_t size = A.width * A.height * sizeof(float);
     hipMalloc(&d_A.elements, size);
     hipMemcpy(d_A.elements, A.elements, size,
                hipMemcpyHostToDevice);
     Matrix d_B;
     d_B.width = d_B.stride = B.width; d_B.height = B.height;
     size = B.width * B.height * sizeof(float);
     hipMalloc(&d_B.elements, size);
     hipMemcpy(d_B.elements, B.elements, size,
     hipMemcpyHostToDevice);

     // Allocate C in device memory
     Matrix d_C;
     d_C.width = d_C.stride = C.width; d_C.height = C.height;
     size = C.width * C.height * sizeof(float);
     hipMalloc(&d_C.elements, size);

     // Invoke kernel
     dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
     dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
     MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

     // Read C from device memory
     hipMemcpy(C.elements, d_C.elements, size,
                hipMemcpyDeviceToHost);

     // Free device memory
     hipFree(d_A.elements);
     hipFree(d_B.elements);
     hipFree(d_C.elements);
 }

 // Matrix multiplication kernel called by MatMul()
  __global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
 {
     // Block row and column
     int blockRow = blockIdx.y;
     int blockCol = blockIdx.x;

     // Each thread block computes one sub-matrix Csub of C
     Matrix Csub = GetSubMatrix(C, blockRow, blockCol);

     // Each thread computes one element of Csub
     // by accumulating results into Cvalue
     float Cvalue = 0;

     // Thread row and column within Csub
     int row = threadIdx.y;
     int col = threadIdx.x;

     // Loop over all the sub-matrices of A and B that are
     // required to compute Csub
     // Multiply each pair of sub-matrices together
     // and accumulate the results
     for (int m = 0; m < (A.width / BLOCK_SIZE); ++m) {

         // Get sub-matrix Asub of A
         Matrix Asub = GetSubMatrix(A, blockRow, m);

         // Get sub-matrix Bsub of B
         Matrix Bsub = GetSubMatrix(B, m, blockCol);

         // Shared memory used to store Asub and Bsub respectively
         __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
         __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

         // Load Asub and Bsub from device memory to shared memory
         // Each thread loads one element of each sub-matrix
         As[row][col] = GetElement(Asub, row, col);
         Bs[row][col] = GetElement(Bsub, row, col);

         // Synchronize to make sure the sub-matrices are loaded
         // before starting the computation
         __syncthreads();
         // Multiply Asub and Bsub together
         for (int e = 0; e < BLOCK_SIZE; ++e)
             Cvalue += As[row][e] * Bs[e][col];

         // Synchronize to make sure that the preceding
         // computation is done before loading two new
         // sub-matrices of A and B in the next iteration
         //其作用是保证 block 内的所有线程都已经运行到调用__syncthreads()的位置
         //保证各个线程看到的存储器是一样的。
         __syncthreads();
     }

     // Write Csub to device memory
     // Each thread writes one element
     SetElement(Csub, row, col, Cvalue);
 }

 void testMatMul(){
	  int width = 1 << 10;
	  int height = 1 << 10;
	  int stride = 1 << 10;
	  Matrix *A, *B, *C;

	  // 申请托管内存
	  hipMallocManaged((void**)&A, sizeof(Matrix));
	  hipMallocManaged((void**)&B, sizeof(Matrix));
	  hipMallocManaged((void**)&C, sizeof(Matrix));
	  int nBytes = width * height * sizeof(float);
	  hipMallocManaged((void**)&A->elements, nBytes);
	  hipMallocManaged((void**)&B->elements, nBytes);
	  hipMallocManaged((void**)&C->elements, nBytes);

	  A->height = height;
	  A->width = width;
	  A->stride = stride;
	  B->height = height;
	  B->width = width;
	  B->stride = stride;
	  C->height = height;
	  C->width = width;
	  C->stride = stride;

	  for (int i = 0; i < width * height; ++i)
	  {
		  A->elements[i] = 1.0;
		  B->elements[i] = 2.0;
	  }

	  MatMul(*A,*B,*C);
	  hipDeviceSynchronize();
	// 检查执行结果
	for(int i=0;i < 10;i++){
		std::cout << C->elements[i] << " ";
	}
	hipFree(A);
	hipFree(B);
	hipFree(C);
  }

